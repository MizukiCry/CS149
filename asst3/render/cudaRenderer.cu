#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans)                                                    \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

struct GlobalConstants {

  SceneName sceneName;

  int numCircles;
  float *position;
  float *velocity;
  float *color;
  float *radius;

  int imageWidth;
  int imageHeight;
  float *imageData;

  float invWidth;
  float invHeight;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int cuConstNoiseYPermutationTable[256];
__constant__ int cuConstNoiseXPermutationTable[256];
__constant__ float cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float cuConstColorRamp[COLOR_MAP_SIZE][3];

// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "circleBoxTest.cu_inl"
#include "lookupColor.cu_inl"
#include "noiseCuda.cu_inl"

// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

  int imageX = blockIdx.x * blockDim.x + threadIdx.x;
  int imageY = blockIdx.y * blockDim.y + threadIdx.y;

  int width = cuConstRendererParams.imageWidth;
  int height = cuConstRendererParams.imageHeight;

  if (imageX >= width || imageY >= height)
    return;

  int offset = 4 * (imageY * width + imageX);
  float shade = .4f + .45f * static_cast<float>(height - imageY) / height;
  float4 value = make_float4(shade, shade, shade, 1.f);

  // write to global memory: As an optimization, I use a float4
  // store, that results in more efficient code than if I coded this
  // up as four seperate fp32 stores.
  *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

  int imageX = blockIdx.x * blockDim.x + threadIdx.x;
  int imageY = blockIdx.y * blockDim.y + threadIdx.y;

  int width = cuConstRendererParams.imageWidth;
  int height = cuConstRendererParams.imageHeight;

  if (imageX >= width || imageY >= height)
    return;

  int offset = 4 * (imageY * width + imageX);
  float4 value = make_float4(r, g, b, a);

  // write to global memory: As an optimization, I use a float4
  // store, that results in more efficient code than if I coded this
  // up as four seperate fp32 stores.
  *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
  const float dt = 1.f / 60.f;
  const float pi = 3.14159;
  const float maxDist = 0.25f;

  float *velocity = cuConstRendererParams.velocity;
  float *position = cuConstRendererParams.position;
  float *radius = cuConstRendererParams.radius;

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= cuConstRendererParams.numCircles)
    return;

  if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update
    return;
  }

  // determine the fire-work center/spark indices
  int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
  int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

  int index3i = 3 * fIdx;
  int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
  int index3j = 3 * sIdx;

  float cx = position[index3i];
  float cy = position[index3i + 1];

  // update position
  position[index3j] += velocity[index3j] * dt;
  position[index3j + 1] += velocity[index3j + 1] * dt;

  // fire-work sparks
  float sx = position[index3j];
  float sy = position[index3j + 1];

  // compute vector from firework-spark
  float cxsx = sx - cx;
  float cysy = sy - cy;

  // compute distance from fire-work
  float dist = sqrt(cxsx * cxsx + cysy * cysy);
  if (dist > maxDist) { // restore to starting position
    // random starting position on fire-work's rim
    float angle = (sfIdx * 2 * pi) / NUM_SPARKS;
    float sinA = sin(angle);
    float cosA = cos(angle);
    float x = cosA * radius[fIdx];
    float y = sinA * radius[fIdx];

    position[index3j] = position[index3i] + x;
    position[index3j + 1] = position[index3i + 1] + y;
    position[index3j + 2] = 0.0f;

    // travel scaled unit length
    velocity[index3j] = cosA / 5.0;
    velocity[index3j + 1] = sinA / 5.0;
    velocity[index3j + 2] = 0.0f;
  }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= cuConstRendererParams.numCircles)
    return;

  float *radius = cuConstRendererParams.radius;

  float cutOff = 0.5f;
  // place circle back in center after reaching threshold radisus
  if (radius[index] > cutOff) {
    radius[index] = 0.02f;
  } else {
    radius[index] += 0.01f;
  }
}

// kernelAdvanceBouncingBalls
//
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() {
  const float dt = 1.f / 60.f;
  const float kGravity = -2.8f; // sorry Newton
  const float kDragCoeff = -0.8f;
  const float epsilon = 0.001f;

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= cuConstRendererParams.numCircles)
    return;

  float *velocity = cuConstRendererParams.velocity;
  float *position = cuConstRendererParams.position;

  int index3 = 3 * index;
  // reverse velocity if center position < 0
  float oldVelocity = velocity[index3 + 1];
  float oldPosition = position[index3 + 1];

  if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition
    return;
  }

  if (position[index3 + 1] < 0 && oldVelocity < 0.f) { // bounce ball
    velocity[index3 + 1] *= kDragCoeff;
  }

  // update velocity: v = u + at (only along y-axis)
  velocity[index3 + 1] += kGravity * dt;

  // update positions (only along y-axis)
  position[index3 + 1] += velocity[index3 + 1] * dt;

  if (fabsf(velocity[index3 + 1] - oldVelocity) < epsilon &&
      oldPosition < 0.0f &&
      fabsf(position[index3 + 1] - oldPosition) < epsilon) { // stop ball
    velocity[index3 + 1] = 0.f;
    position[index3 + 1] = 0.f;
  }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= cuConstRendererParams.numCircles)
    return;

  const float dt = 1.f / 60.f;
  const float kGravity = -1.8f; // sorry Newton
  const float kDragCoeff = 2.f;

  int index3 = 3 * index;

  float *positionPtr = &cuConstRendererParams.position[index3];
  float *velocityPtr = &cuConstRendererParams.velocity[index3];

  // loads from global memory
  float3 position = *((float3 *)positionPtr);
  float3 velocity = *((float3 *)velocityPtr);

  // hack to make farther circles move more slowly, giving the
  // illusion of parallax
  float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

  // add some noise to the motion to make the snow flutter
  float3 noiseInput;
  noiseInput.x = 10.f * position.x;
  noiseInput.y = 10.f * position.y;
  noiseInput.z = 255.f * position.z;
  float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
  noiseForce.x *= 7.5f;
  noiseForce.y *= 5.f;

  // drag
  float2 dragForce;
  dragForce.x = -1.f * kDragCoeff * velocity.x;
  dragForce.y = -1.f * kDragCoeff * velocity.y;

  // update positions
  position.x += velocity.x * dt;
  position.y += velocity.y * dt;

  // update velocities
  velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
  velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

  float radius = cuConstRendererParams.radius[index];

  // if the snowflake has moved off the left, right or bottom of
  // the screen, place it back at the top and give it a
  // pseudorandom x position and velocity.
  if ((position.y + radius < 0.f) || (position.x + radius) < -0.f ||
      (position.x - radius) > 1.f) {
    noiseInput.x = 255.f * position.x;
    noiseInput.y = 255.f * position.y;
    noiseInput.z = 255.f * position.z;
    noiseForce = cudaVec2CellNoise(noiseInput, index);

    position.x = .5f + .5f * noiseForce.x;
    position.y = 1.35f + radius;

    // restart from 0 vertical velocity.  Choose a
    // pseudo-random horizontal velocity.
    velocity.x = 2.f * noiseForce.y;
    velocity.y = 0.f;
  }

  // store updated positions and velocities to global memory
  *((float3 *)positionPtr) = position;
  *((float3 *)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void shadePixel(int circleIndex, float2 pixelCenter,
                                      float3 p, float4 *imagePtr) {

  float diffX = p.x - pixelCenter.x;
  float diffY = p.y - pixelCenter.y;
  float pixelDist = diffX * diffX + diffY * diffY;

  float rad = cuConstRendererParams.radius[circleIndex];
  ;
  float maxDist = rad * rad;

  // circle does not contribute to the image
  if (pixelDist > maxDist)
    return;

  float3 rgb;
  float alpha;

  // there is a non-zero contribution.  Now compute the shading value

  // suggestion: This conditional is in the inner loop.  Although it
  // will evaluate the same for all threads, there is overhead in
  // setting up the lane masks etc to implement the conditional.  It
  // would be wise to perform this logic outside of the loop next in
  // kernelRenderCircles.  (If feeling good about yourself, you
  // could use some specialized template magic).
  if (cuConstRendererParams.sceneName == SNOWFLAKES ||
      cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

    const float kCircleMaxAlpha = .5f;
    const float falloffScale = 4.f;

    float normPixelDist = sqrt(pixelDist) / rad;
    rgb = lookupColor(normPixelDist);

    float maxAlpha = .6f + .4f * (1.f - p.z);
    maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f),
                                       0.f); // kCircleMaxAlpha * clamped value
    alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

  } else {
    // simple: each circle has an assigned color
    int index3 = 3 * circleIndex;
    rgb = *(float3 *)&(cuConstRendererParams.color[index3]);
    alpha = .5f;
  }

  float oneMinusAlpha = 1.f - alpha;

  // BEGIN SHOULD-BE-ATOMIC REGION
  // global memory read

  float4 existingColor = *imagePtr;
  float4 newColor;
  newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
  newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
  newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
  newColor.w = alpha + existingColor.w;

  // global memory write
  *imagePtr = newColor;

  // END SHOULD-BE-ATOMIC REGION
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles() {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= cuConstRendererParams.numCircles)
    return;

  int index3 = 3 * index;

  // read position and radius
  float3 p = *(float3 *)(&cuConstRendererParams.position[index3]);
  float rad = cuConstRendererParams.radius[index];

  // compute the bounding box of the circle. The bound is in integer
  // screen coordinates, so it's clamped to the edges of the screen.
  short imageWidth = cuConstRendererParams.imageWidth;
  short imageHeight = cuConstRendererParams.imageHeight;
  short minX = static_cast<short>(imageWidth * (p.x - rad));
  short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
  short minY = static_cast<short>(imageHeight * (p.y - rad));
  short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

  // a bunch of clamps.  Is there a CUDA built-in for this?
  short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
  short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
  short screenMinY =
      (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
  short screenMaxY =
      (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

  float invWidth = 1.f / imageWidth;
  float invHeight = 1.f / imageHeight;

  // for all pixels in the bonding box
  for (int pixelY = screenMinY; pixelY < screenMaxY; pixelY++) {
    float4 *imgPtr =
        (float4 *)(&cuConstRendererParams
                        .imageData[4 * (pixelY * imageWidth + screenMinX)]);
    for (int pixelX = screenMinX; pixelX < screenMaxX; pixelX++) {
      float2 pixelCenterNorm =
          make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                      invHeight * (static_cast<float>(pixelY) + 0.5f));
      shadePixel(index, pixelCenterNorm, p, imgPtr);
      imgPtr++;
    }
  }
}

namespace Solution1 {
__global__ void kernelRenderPixelsWithCircles(int MinX, int MaxX, int MinY,
                                              int MaxY, int numCircles,
                                              int *Circles) {
  int pixelX = MinX + blockIdx.x * blockDim.x + threadIdx.x;
  int pixelY = MinY + blockIdx.y * blockDim.y + threadIdx.y;

  if (pixelX >= MaxX || pixelY >= MaxY)
    return;

  float2 pixelCenterNorm =
      make_float2((pixelX + 0.5) / cuConstRendererParams.imageWidth,
                  (pixelY + 0.5) / cuConstRendererParams.imageHeight);

  float4 *imgPtr = reinterpret_cast<float4 *>(
      &cuConstRendererParams
           .imageData[4 *
                      (pixelY * cuConstRendererParams.imageWidth + pixelX)]);

  for (int i = 0; i < numCircles; i++) {
    int index3 = 3 * Circles[i];
    shadePixel(
        Circles[i], pixelCenterNorm,
        *reinterpret_cast<float3 *>(&cuConstRendererParams.position[index3]),
        imgPtr);
  }
}

void renderPixelsWithCircles(int MinX, int MaxX, int MinY, int MaxY,
                             int numCircles, int *Circles) {
  // printf("renderPixelsWithCircles\n");
  // printf("MinX: %d, MaxX: %d, MinY: %d, MaxY: %d\n", MinX, MaxX, MinY, MaxY);
  // printf("numCircles: %d\n", numCircles);
  // for (int i = 0; i < numCircles; i++) {
  //   printf("Circles[%d]: %d\n", i, Circles[i]);
  // }
  if (numCircles == 0) {
    return;
  }

  static constexpr int THREADS_PER_BLOCK = 256;
  int blockDimX = std::min(THREADS_PER_BLOCK, MaxX - MinX);
  int blockDimY = std::min(THREADS_PER_BLOCK / blockDimX, MaxY - MinY);
  dim3 blockDim(blockDimX, blockDimY);

  int gridDimX = (MaxX - MinX + blockDim.x - 1) / blockDim.x;
  int gridDimY = (MaxY - MinY + blockDim.y - 1) / blockDim.y;
  dim3 gridDim(gridDimX, gridDimY);

  // printf("blockDim: %d, %d\n", blockDim.x, blockDim.y);
  // printf("gridDim: %d, %d\n", gridDim.x, gridDim.y);

  kernelRenderPixelsWithCircles<<<gridDim, blockDim>>>(MinX, MaxX, MinY, MaxY,
                                                       numCircles, Circles);
}

void renderPixelsWithAllCircles(int numCircles, int width, int height) {
  int *Circles;
  hipMallocManaged(&Circles, numCircles * sizeof(int));
  for (int i = 0; i < numCircles; i++) {
    Circles[i] = i;
  }

  renderPixelsWithCircles(0, width, 0, height, numCircles, Circles);
  hipDeviceSynchronize();

  hipFree(Circles);
}

constexpr int BLOCK_SIZE = 16;

__global__ void kernelGetCirclesInBlock(int *numBlockCircles, int *blockCircles,
                                        int width, int height, int numBlocksX) {
  int threadX = blockIdx.x * blockDim.x + threadIdx.x;
  int threadY = blockIdx.y * blockDim.y + threadIdx.y;
  int blockMinX = threadX * BLOCK_SIZE;
  int blockMinY = threadY * BLOCK_SIZE;

  if (blockMinX >= width || blockMinY >= height)
    return;

  int threadID = threadY * numBlocksX + threadX;
  int blockMaxX = min(blockMinX + BLOCK_SIZE, width);
  int blockMaxY = min(blockMinY + BLOCK_SIZE, height);

  float invWidth = 1.f / width;
  float invHeight = 1.f / height;
  float boxL = blockMinX * invWidth;
  float boxR = blockMaxX * invWidth;
  float boxT = blockMinY * invHeight;
  float boxB = blockMaxY * invHeight;

  numBlockCircles[threadID] = 0;
  int *ptr = blockCircles + threadID * cuConstRendererParams.numCircles;
  for (int i = 0; i < cuConstRendererParams.numCircles; i++) {
    int index3 = 3 * i;
    if (circleInBox(cuConstRendererParams.position[index3],
                    cuConstRendererParams.position[index3 + 1],
                    cuConstRendererParams.radius[i], boxL, boxR, boxT, boxB)) {
      ptr[numBlockCircles[threadID]++] = i;
    }
  }

  // printf("Thread: %d, %d, %d numBlockCircles: %d\n", threadX, threadY,
  // threadID,
  //        numBlockCircles[threadID]);
}

void renderBlockedPixelsWithAllCircles(int numCircles, int width, int height) {
  // printf("renderBlockedPixelsWithAllCircles\n");
  // printf("numCircles: %d, width: %d, height: %d\n", numCircles, width,
  // height);

  int numBlocksX = (width + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int numBlocksY = (height + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int numBlocks = numBlocksX * numBlocksY;

  // printf("numBlocksX: %d, numBlocksY: %d\n", numBlocksX, numBlocksY);
  // printf("numBlocks: %d\n", numBlocks);

  int *numBlockCircles;
  cudaCheckError(hipMallocManaged(&numBlockCircles, numBlocks * sizeof(int)));
  int *blockCircles;
  cudaCheckError(
      hipMallocManaged(&blockCircles, numBlocks * numCircles * sizeof(int)));

  dim3 gridDim((numBlocksX + BLOCK_SIZE - 1) / BLOCK_SIZE,
               (numBlocksY + BLOCK_SIZE - 1) / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

  // printf("gridDim: %d, %d\n", gridDim.x, gridDim.y);
  // printf("blockDim: %d, %d\n", blockDim.x, blockDim.y);

  kernelGetCirclesInBlock<<<gridDim, blockDim>>>(numBlockCircles, blockCircles,
                                                 width, height, numBlocksX);
  hipDeviceSynchronize();
  // printf("kernelGetCirclesInBlock finished\n");

  for (int i = 0; i < numBlocksX; i++) {
    for (int j = 0; j < numBlocksY; j++) {
      int blockIndex = j * numBlocksX + i;
      int blockMinX = i * BLOCK_SIZE;
      int blockMinY = j * BLOCK_SIZE;
      int blockMaxX = std::min(blockMinX + BLOCK_SIZE, width);
      int blockMaxY = std::min(blockMinY + BLOCK_SIZE, height);
      // printf("blockIndex: %d, blockMinX: %d, blockMaxX: %d, blockMinY: %d, "
      //        "blockMaxY: %d\n",
      //        blockIndex, blockMinX, blockMaxX, blockMinY, blockMaxY);
      // printf("numBlockCircles: %d\n", numBlockCircles[blockIndex]);
      renderPixelsWithCircles(blockMinX, blockMaxX, blockMinY, blockMaxY,
                              numBlockCircles[blockIndex],
                              blockCircles + blockIndex * numCircles);
      hipDeviceSynchronize();
    }
  }
  hipDeviceSynchronize();

  hipFree(blockCircles);
  hipFree(numBlockCircles);
}
} // namespace Solution1

#include <thrust/scan.h>
#include <thrust/sort.h>

namespace Solution2 {

constexpr int THREADS_PER_BLOCK = 256;
constexpr int BLOCK_SIZE = 16;

__inline__ __device__ int clamp(int x, int low, int high) {
  return x > low ? (x < high ? x : high) : low;
}

template <typename T> T readKernel(const T *ptr) {
  T value;
  hipMemcpy(&value, ptr, sizeof(T), hipMemcpyDeviceToHost);
  return value;
}

__inline__ __device__ bool pixelInCircle(float2 pixelCenter, float3 p,
                                         float rad) {
  float dx = p.x - pixelCenter.x;
  float dy = p.y - pixelCenter.y;
  return dx * dx + dy * dy <= rad * rad;
}

// for each circle, calculate the bounding box of the circle
__global__ void kernelGetCirclesSize(int *circlesSizePrefixSum, short *minX,
                                     short *maxX, short *minY, short *maxY) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= cuConstRendererParams.numCircles)
    return;

  int index3 = 3 * index;
  float3 p =
      *reinterpret_cast<float3 *>(&cuConstRendererParams.position[index3]);
  float rad = cuConstRendererParams.radius[index];

  short imageWidth = cuConstRendererParams.imageWidth;
  short imageHeight = cuConstRendererParams.imageHeight;

  minX[index] = clamp(imageWidth * (p.x - rad), 0, imageWidth);
  maxX[index] = clamp(imageWidth * (p.x + rad) + 1, 0, imageWidth);
  minY[index] = clamp(imageHeight * (p.y - rad), 0, imageHeight);
  maxY[index] = clamp(imageHeight * (p.y + rad) + 1, 0, imageHeight);

  circlesSizePrefixSum[index] =
      (maxX[index] - minX[index]) * (maxY[index] - minY[index]);
}

// for each circle, calculate the pixels covered by the circle
__global__ void kernelGetCirclesPixels(int *pixelsId, int *circlesId,
                                       int *circlesSizePrefixSum, short minX,
                                       short maxX, short minY, short maxY,
                                       int index) {
  int pixelX = minX + blockIdx.x * blockDim.x + threadIdx.x;
  int pixelY = minY + blockIdx.y * blockDim.y + threadIdx.y;

  if (pixelX >= maxX || pixelY >= maxY)
    return;

  int globalOffset = index == 0 ? 0 : circlesSizePrefixSum[index - 1];

  int offset = (pixelY - minY) * (maxX - minX) + (pixelX - minX);
  int pixelId = pixelY * cuConstRendererParams.imageWidth + pixelX;

  if (pixelInCircle(
          make_float2((pixelX + 0.5) / cuConstRendererParams.imageWidth,
                      (pixelY + 0.5) / cuConstRendererParams.imageHeight),
          *reinterpret_cast<float3 *>(
              &cuConstRendererParams.position[3 * index]),
          cuConstRendererParams.radius[index])) {
    pixelsId[globalOffset + offset] = pixelId;
    circlesId[globalOffset + offset] = index;
  } else {
    pixelsId[globalOffset + offset] = -1;
    circlesId[globalOffset + offset] = -1;
  }
}

// __device__ void kernelBlendPixel() {}

__global__ void kernelRenderPixels(int *pixelsId, int *circlesId,
                                   int totalCirclesSize) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= totalCirclesSize)
    return;

  if (pixelsId[index] == -1 ||
      (index > 0 && pixelsId[index] == pixelsId[index - 1]))
    return;

  while (true) {
    int pixelX = pixelsId[index] % cuConstRendererParams.imageWidth;
    int pixelY = pixelsId[index] / cuConstRendererParams.imageWidth;

    shadePixel(circlesId[index],
               make_float2((pixelX + 0.5) / cuConstRendererParams.imageWidth,
                           (pixelY + 0.5) / cuConstRendererParams.imageHeight),
               *reinterpret_cast<float3 *>(
                   &cuConstRendererParams.position[3 * circlesId[index]]),
               reinterpret_cast<float4 *>(
                   &cuConstRendererParams.imageData[4 * pixelsId[index]]));

    if (index + 1 < totalCirclesSize &&
        pixelsId[index] == pixelsId[index + 1]) {
      ++index;
    } else {
      break;
    }
  }
}

void renderCircles(int numCircles) {
  int *circlesSizePrefixSum;
  short *minX, *maxX, *minY, *maxY;
  cudaCheckError(hipMalloc(&circlesSizePrefixSum, numCircles * sizeof(int)));
  cudaCheckError(hipMalloc(&minX, numCircles * sizeof(short)));
  cudaCheckError(hipMalloc(&maxX, numCircles * sizeof(short)));
  cudaCheckError(hipMalloc(&minY, numCircles * sizeof(short)));
  cudaCheckError(hipMalloc(&maxY, numCircles * sizeof(short)));

  kernelGetCirclesSize<<<(numCircles + THREADS_PER_BLOCK - 1) /
                             THREADS_PER_BLOCK,
                         THREADS_PER_BLOCK>>>(circlesSizePrefixSum, minX, maxX,
                                              minY, maxY);
  hipDeviceSynchronize();

  thrust::inclusive_scan(thrust::device, circlesSizePrefixSum,
                         circlesSizePrefixSum + numCircles,
                         circlesSizePrefixSum);

  // pixel covered by circles
  int *pixelsId, *circlesId;
  int totalCirclesSize = readKernel(&circlesSizePrefixSum[numCircles - 1]);
  cudaCheckError(hipMalloc(&pixelsId, totalCirclesSize * sizeof(int)));
  cudaCheckError(hipMalloc(&circlesId, totalCirclesSize * sizeof(int)));

  short *minXHost = new short[numCircles];
  short *maxXHost = new short[numCircles];
  short *minYHost = new short[numCircles];
  short *maxYHost = new short[numCircles];

  cudaCheckError(hipMemcpy(minXHost, minX, numCircles * sizeof(short),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(maxXHost, maxX, numCircles * sizeof(short),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(minYHost, minY, numCircles * sizeof(short),
                            hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpy(maxYHost, maxY, numCircles * sizeof(short),
                            hipMemcpyDeviceToHost));

  for (int i = 0; i < numCircles; i++) {
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((maxXHost[i] - minXHost[i] + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (maxYHost[i] - minYHost[i] + BLOCK_SIZE - 1) / BLOCK_SIZE);
    kernelGetCirclesPixels<<<gridDim, blockDim>>>(
        pixelsId, circlesId, circlesSizePrefixSum, minXHost[i], maxXHost[i],
        minYHost[i], maxYHost[i], i);
  }
  hipDeviceSynchronize();

  thrust::stable_sort_by_key(thrust::device, pixelsId,
                             pixelsId + totalCirclesSize, circlesId);

  kernelRenderPixels<<<(totalCirclesSize + THREADS_PER_BLOCK - 1) /
                           THREADS_PER_BLOCK,
                       THREADS_PER_BLOCK>>>(pixelsId, circlesId,
                                            totalCirclesSize);
  hipDeviceSynchronize();

  delete[] minXHost;
  delete[] maxXHost;
  delete[] minYHost;
  delete[] maxYHost;

  hipFree(circlesSizePrefixSum);
  hipFree(minX);
  hipFree(maxX);
  hipFree(minY);
  hipFree(maxY);
  hipFree(pixelsId);
  hipFree(circlesId);
}
} // namespace Solution2

namespace Solution3 {

constexpr int BLOCK_DIM = 16;
constexpr int BLOCK_SIZE = BLOCK_DIM * BLOCK_DIM;

#define SCAN_BLOCK_DIM BLOCK_SIZE
#include "exclusiveScan.cu_inl"

__global__ void kernelRenderCircles() {
  __shared__ uint circleIsInBox[BLOCK_SIZE];
  __shared__ uint circleIndex[BLOCK_SIZE];
  __shared__ uint scratch[2 * BLOCK_SIZE];
  __shared__ int inBoxCircles[BLOCK_SIZE];

  int boxL = blockIdx.x * BLOCK_DIM;
  int boxB = blockIdx.y * BLOCK_DIM;
  int boxR = min(boxL + BLOCK_DIM, cuConstRendererParams.imageWidth);
  int boxT = min(boxB + BLOCK_DIM, cuConstRendererParams.imageHeight);
  float boxLNorm = boxL * cuConstRendererParams.invWidth;
  float boxRNorm = boxR * cuConstRendererParams.invWidth;
  float boxTNorm = boxT * cuConstRendererParams.invHeight;
  float boxBNorm = boxB * cuConstRendererParams.invHeight;

  int index = threadIdx.y * BLOCK_DIM + threadIdx.x;
  int pixelX = boxL + threadIdx.x;
  int pixelY = boxB + threadIdx.y;
  int pixelId = pixelY * cuConstRendererParams.imageWidth + pixelX;

  for (int i = 0; i < cuConstRendererParams.numCircles; i += BLOCK_SIZE) {
    int circleId = i + index;
    if (circleId < cuConstRendererParams.numCircles) {
      float3 p = *reinterpret_cast<float3 *>(
          &cuConstRendererParams.position[3 * circleId]);
      circleIsInBox[index] =
          circleInBox(p.x, p.y, cuConstRendererParams.radius[circleId],
                      boxLNorm, boxRNorm, boxTNorm, boxBNorm);
    } else {
      circleIsInBox[index] = 0;
    }
    __syncthreads();

    sharedMemExclusiveScan(index, circleIsInBox, circleIndex, scratch,
                           BLOCK_SIZE);
    if (circleIsInBox[index]) {
      inBoxCircles[circleIndex[index]] = circleId;
    }
    __syncthreads();

    int numCirclesInBox =
        circleIndex[BLOCK_SIZE - 1] + circleIsInBox[BLOCK_SIZE - 1];
    __syncthreads();

    if (pixelX < boxR && pixelY < boxT) {
      float4 *imgPtr = reinterpret_cast<float4 *>(
          &cuConstRendererParams.imageData[4 * pixelId]);
      for (int j = 0; j < numCirclesInBox; j++) {
        circleId = inBoxCircles[j];
        shadePixel(
            circleId,
            make_float2((pixelX + 0.5) * cuConstRendererParams.invWidth,
                        (pixelY + 0.5) * cuConstRendererParams.invHeight),
            *reinterpret_cast<float3 *>(
                &cuConstRendererParams.position[3 * circleId]),
            imgPtr);
      }
    }
  }
}

void renderCircles(int width, int height) {
  kernelRenderCircles<<<dim3((width + BLOCK_DIM - 1) / BLOCK_DIM,
                             (height + BLOCK_DIM - 1) / BLOCK_DIM),
                        dim3(BLOCK_DIM, BLOCK_DIM)>>>();
  cudaCheckError(hipDeviceSynchronize());
}
} // namespace Solution3

////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
  image = NULL;

  numCircles = 0;
  position = NULL;
  velocity = NULL;
  color = NULL;
  radius = NULL;

  cudaDevicePosition = NULL;
  cudaDeviceVelocity = NULL;
  cudaDeviceColor = NULL;
  cudaDeviceRadius = NULL;
  cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

  if (image) {
    delete image;
  }

  if (position) {
    delete[] position;
    delete[] velocity;
    delete[] color;
    delete[] radius;
  }

  if (cudaDevicePosition) {
    hipFree(cudaDevicePosition);
    hipFree(cudaDeviceVelocity);
    hipFree(cudaDeviceColor);
    hipFree(cudaDeviceRadius);
    hipFree(cudaDeviceImageData);
  }
}

const Image *CudaRenderer::getImage() {

  // need to copy contents of the rendered image from device memory
  // before we expose the Image object to the caller

  printf("Copying image data from device\n");

  hipMemcpy(image->data, cudaDeviceImageData,
             sizeof(float) * 4 * image->width * image->height,
             hipMemcpyDeviceToHost);

  return image;
}

void CudaRenderer::loadScene(SceneName scene) {
  sceneName = scene;
  loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void CudaRenderer::setup() {

  int deviceCount = 0;
  std::string name;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Initializing CUDA for CudaRenderer\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    name = deviceProps.name;

    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");

  // By this time the scene should be loaded.  Now copy all the key
  // data structures into device memory so they are accessible to
  // CUDA kernels
  //
  // See the CUDA Programmer's Guide for descriptions of
  // hipMalloc and hipMemcpy

  hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
  hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
  hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
  hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
  hipMalloc(&cudaDeviceImageData,
             sizeof(float) * 4 * image->width * image->height);

  hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles,
             hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles,
             hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles,
             hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles,
             hipMemcpyHostToDevice);

  // Initialize parameters in constant memory.  We didn't talk about
  // constant memory in class, but the use of read-only constant
  // memory here is an optimization over just sticking these values
  // in device global memory.  NVIDIA GPUs have a few special tricks
  // for optimizing access to constant memory.  Using global memory
  // here would have worked just as well.  See the Programmer's
  // Guide for more information about constant memory.

  GlobalConstants params;
  params.sceneName = sceneName;
  params.numCircles = numCircles;
  params.imageWidth = image->width;
  params.imageHeight = image->height;
  params.position = cudaDevicePosition;
  params.velocity = cudaDeviceVelocity;
  params.color = cudaDeviceColor;
  params.radius = cudaDeviceRadius;
  params.imageData = cudaDeviceImageData;
  params.invWidth = 1.f / image->width;
  params.invHeight = 1.f / image->height;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

  // also need to copy over the noise lookup tables, so we can
  // implement noise on the GPU
  int *permX;
  int *permY;
  float *value1D;
  getNoiseTables(&permX, &permY, &value1D);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

  // last, copy over the color table that's used by the shading
  // function for circles in the snowflake demo

  float lookupTable[COLOR_MAP_SIZE][3] = {
      {1.f, 1.f, 1.f}, {1.f, 1.f, 1.f},  {.8f, .9f, 1.f},
      {.8f, .9f, 1.f}, {.8f, 0.8f, 1.f},
  };

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable,
                     sizeof(float) * 3 * COLOR_MAP_SIZE);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height) {

  if (image)
    delete image;
  image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage() {

  // 256 threads per block is a healthy number
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x,
               (image->height + blockDim.y - 1) / blockDim.y);

  if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
    kernelClearImageSnowflake<<<gridDim, blockDim>>>();
  } else {
    kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
  }
  hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void CudaRenderer::advanceAnimation() {
  // 256 threads per block is a healthy number
  dim3 blockDim(256, 1);
  dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

  // only the snowflake scene has animation
  if (sceneName == SNOWFLAKES) {
    kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
  } else if (sceneName == BOUNCING_BALLS) {
    kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
  } else if (sceneName == HYPNOSIS) {
    kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
  } else if (sceneName == FIREWORKS) {
    kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
  }
  hipDeviceSynchronize();
}

void CudaRenderer::render() {
  // 256 threads per block is a healthy number
  // dim3 blockDim(256, 1);
  // dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);
  // kernelRenderCircles<<<gridDim, blockDim>>>();
  // hipDeviceSynchronize();

  Solution3::renderCircles(image->width, image->height);
}
